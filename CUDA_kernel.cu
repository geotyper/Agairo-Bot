#include "hip/hip_runtime.h"
﻿#pragma once
#include "Cuda_kernel.cuh"


__device__ float anglebetween2vectors(float x, float y, float x2, float y2)
{

	float arc = (x * x + y * y) * (x2 * x2 + y2 * y2);

	arc = sqrtf(arc);
	if (arc > 0.f)
	{
		arc = acosf((x * x2 + y * y2) / arc);
		if ((x * y2 - y * x2) < 0.f)
			arc = -arc;
	}
	return arc;
}


__device__  float Sigmoid(float x)
{
	if (x < -10.0f) return 0.0f;
	else if (x > 10.0f) return 1.0f;
	return (float)(1.0f / (1.0f + expf(-x)));
}

__device__  float Tanh(float x)
{
	if (x < -10.0f) return -1.0f;
	else if (x > 10.0f) return 1.0f;
	return (float)(tanhf(x));
}


__device__  float SoftSignFunction(float xValue)
{
	return xValue / (1.f + fabs(xValue));
}


__device__ void RndGen(float *Gen, int mGl, int nGl, float *out1, int bot, int numberLayers, int layer, int LayerDim)
{
	int rows = mGl; int cols = mGl;

	for (int i = 0; i < rows; ++i)
		for (int j = 0; j < cols; ++j)
			out1[((bot*numberLayers + layer)*LayerDim + i)*LayerDim + j] = 0;


}


__host__ void InitBotList(std::vector<Bot>& BotList, std::vector<float>& Sector, Constant& ct)
{
	std::list <Bot> ::iterator it;
	std::list<Bot> ServerInitBots;

	BotList.clear();
	Sector.clear();

	int botId = 0;

	for (int d = 0; d < ct.Depth; d++)
	{
		for (int i = 0; i < ct.BotPopulationSize; i++)
		{
			int deltastep = 30;

			Bot bot;

			bot.intID = botId;
			bot.TTF = 1;
			bot.Mass = 57;
			bot.Depth = d;
			bot.IntType = 1;
			bot.Xvelocity = 0;
			bot.Yvelocity = 0;
			bot.color = i + 1;

			bot.angle = (rand() % 360) * 3.141592f / 180;
			bot.speed = (float)rand() / ((float)RAND_MAX + 1);
			bot.Radius = (int)(2 * sqrt(bot.Mass));

			bot.X = rand() % (ct.GAME_WIDTH - 2 *  deltastep) + deltastep;
			bot.Y = rand() % (ct.GAME_HEIGHT - 2 * deltastep) + deltastep;
			/*
			while (ServerInitBots.Exists(s = > s.Depth == d&& Math.Abs(s.X - bot.X) < 55 && Math.Abs(s.Y - bot.Y) < 55))
			{
			bot.X = rand() % (0 + deltastep) + (conworld.GAME_WIDTH - deltastep);
			bot.Y = rand() % (0 + deltastep) + (conworld.GAME_HEIGHT - deltastep);
			}
			*/

			//list <Bot> ::iterator it1;
			bool find_element = true;
			int findElementCount = 0;

			while (find_element)
			{

				std::list <Bot> ::iterator it3;
				for (it3 = ServerInitBots.begin(); it3 != ServerInitBots.end(); ++it3)
				{

					if (it3->Depth == d && abs(it3->X - bot.X) < 75 && abs(it3->Y - bot.Y) < 75)
					{
						bot.X = rand() % (ct.GAME_WIDTH - 2 * deltastep) + deltastep;
						bot.Y = rand() % (ct.GAME_HEIGHT - 2 * deltastep) + deltastep;
						++findElementCount;

					}


				}

				if (findElementCount == 0)
				{
					find_element = false;
				}
				else
				{
					findElementCount = 0;
				}

			}

			bot.xPOI = bot.X;
			bot.yPOI = bot.Y;

			ServerInitBots.push_back(bot);
			botId++;


		}

	//	std::cout << "Number of bots " << ct.BotPopulationSize << " generated in Depth " << d << std::endl;

	}

	//std::cout << "Number of all bots " << ServerInitBots.size() << " generated in all Depth " << ct.Depth << std::endl;

	std::list <Bot> ::iterator it1;
	for (it1 = ServerInitBots.begin(); it1 != ServerInitBots.end(); ++it1)
	{
		BotList.push_back(*it1);
	}

	ServerInitBots.clear();

}

__host__ void InitFoodList(std::vector<Food>& FoodList,  Constant & ct)
{
	FoodList.clear();

	std::list<Food> ServerInitFood;
	std::vector<Food> ServerFoodList;
	int FoodId = 0;


	int botId = 0;

	for (int d = 0; d < ct.Depth; d++)
	{
		for (int i = 0; i < ct.FoodPopulationSize; i++)
		{
			int deltastep = 10;

			Food food;

			//bot.ID[3] = "b";
			food.ID = FoodId;
			food.Mass = ct.FOOD_MASS;
			food.Radius = 2.5f;
			food.IntType = 3;
			food.Depth = d;

			food.X = rand() % (ct.GAME_WIDTH - 2 * deltastep) + deltastep;
			food.Y = rand() % (ct.GAME_HEIGHT - 2 * deltastep) + deltastep;

			bool find_element = true;
			int findElementCount = 0;

			ServerInitFood.push_back(food);
			FoodId++;

			Food food1;

			food1.ID = FoodId;
			food1.Mass = ct.FOOD_MASS;
			food1.Radius = 2.5f;
			food1.IntType = 3;
			food1.Depth = d;
			food1.X = ct.GAME_WIDTH - food.X;
			food1.Y = food.Y;

			ServerInitFood.push_back(food1);
			FoodId++;

			Food food2;

			food2.ID = FoodId;
			food2.Mass = ct.FOOD_MASS;
			food2.Radius = 2.5f;
			food2.IntType = 3;
			food2.Depth = d;
			food2.X = food.X;
			food2.Y = ct.GAME_HEIGHT - food.Y;

			ServerInitFood.push_back(food2);
			FoodId++;

			Food food3;

			food3.ID = FoodId;
			food3.Mass = ct.FOOD_MASS;
			food3.Radius = 2.5f;
			food3.IntType = 3;
			food3.Depth = d;
			food3.X = ct.GAME_WIDTH - food.X;
			food3.Y = ct.GAME_HEIGHT - food.Y;

			ServerInitFood.push_back(food3);
			FoodId++;

		}

		//std::cout << "Number of food " << ct.FoodPopulationSize << " generated in Depth " << d << std::endl;

	}

	//std::cout << "Number of all food " << ServerInitFood.size() << " generated in all Depth " <<ct.Depth << std::endl;

	std::list <Food> ::iterator it;
	for (it = ServerInitFood.begin(); it != ServerInitFood.end(); ++it)
	{
		FoodList.push_back(*it);
	}

	ServerInitFood.clear();


}

__host__ void InitAddFoodList( std::vector<Food>& AddFoodList, Constant & ct)
{
	
	AddFoodList.clear();
	
	std::list<Food> ServerAddInitFood;
	
	int FoodId = 0;

	int botId = 0;

	for (int d = 0; d < ct.Depth; d++)
	{
		for (int i = 0; i < ct.FoodAddPopulationSize; i++)
		{
			int deltastep = 10;

			Food food;

			//bot.ID[3] = "b";
			food.ID = FoodId;
			food.Mass =ct.FOOD_MASS;
			food.Radius = 2.5f;
			food.IntType = 3;
			food.Depth = d;

			food.X = rand() % (ct.GAME_WIDTH - 2 * deltastep) + deltastep;
			food.Y = rand() % (ct.GAME_HEIGHT - 2 * deltastep) + deltastep;

			bool find_element = true;
			int findElementCount = 0;
			
			ServerAddInitFood.push_back(food);
			FoodId++;

			Food food1;

			food1.ID = FoodId;
			food1.Mass = ct.FOOD_MASS;
			food1.Radius = 2.5f;
			food1.IntType = 3;
			food1.Depth = d;
			food1.X = ct.GAME_WIDTH - food.X;
			food1.Y = food.Y;

			ServerAddInitFood.push_back(food1);
			FoodId++;

			Food food2;

			food2.ID = FoodId;
			food2.Mass = ct.FOOD_MASS;
			food2.Radius = 2.5f;
			food2.IntType = 3;
			food2.Depth = d;
			food2.X = food.X;
			food2.Y =ct.GAME_HEIGHT - food.Y;

			ServerAddInitFood.push_back(food2);
			FoodId++;

			Food food3;

			food3.ID = FoodId;
			food3.Mass = ct.FOOD_MASS;
			food3.Radius = 2.5f;
			food3.IntType = 3;
			food3.Depth = d;
			food3.X = ct.GAME_WIDTH - food.X;
			food3.Y = ct.GAME_HEIGHT - food.Y;

			ServerAddInitFood.push_back(food3);
			FoodId++;

		}

		//std::cout << "Number of Add food " << ct.FoodAddPopulationSize << " generated in Depth " << d << std::endl;

	}

	//std::cout << "Number of all Add food " << ServerAddInitFood.size() << " generated in all Depth " << ct.Depth << std::endl;

	std::list <Food> ::iterator it;
	for (it = ServerAddInitFood.begin(); it != ServerAddInitFood.end(); ++it)
	{
		AddFoodList.push_back(*it);
	}

	ServerAddInitFood.clear();

}



__global__ void cudaGARNN2(Bot *bot, int *fitnessIndex, float *rnnDNA, argumentsRNN *RNN, int *Topology, int *TopologyRNN, int NNslide,
	ConstantStruct *Const, int numElements, int gameTick, int middleFitness, int fitness5, int sumFitness)
{
	int  tid = blockIdx.x * blockDim.x + threadIdx.x;
	int  threadN = gridDim.x * blockDim.x;

	int TopologySize = Const->TopologySize;
	int ArrayDim = Const->ArrayDim;


	float crossValueBig = 0.75f;
	float crossValueSmall = 0.35f;


	for (int pos = tid; pos < numElements; pos += threadN)
	{

		if (bot[pos].Fitness<middleFitness)
		{
			if(fabs(rnnDNA[pos*numElements + 175]) < 0.65)
			{ 

					for (int layerT = 0; layerT < Const->TopologySize - 1; layerT++)
					{
						for (int it = 0; it < Topology[layerT] + 1; it++)
						{
							for (int jt = 0; jt < Topology[layerT + 1]; jt++)
							{
								RNN->NNweights[((pos*Const->TopologySize + layerT)*ArrayDim + it)*ArrayDim + jt] =
									rnnDNA[((pos*Const->TopologySize + layerT)*ArrayDim + it)*ArrayDim + jt];

							}

						}

					}

					for (int layerR = 0; layerR < Const->TopologyRNNSize - 1; layerR++)
					{
						for (int ir = 0; ir < TopologyRNN[layerR] + 1; ir++)
						{
							for (int jr = 0; jr < TopologyRNN[layerR]; jr++)
							{
								RNN->MNweights[((pos*Const->TopologySize + layerR)*ArrayDim + ir)*ArrayDim + jr] =
									rnnDNA[NNslide + ((pos*Const->TopologySize + layerR)*ArrayDim + ir)*ArrayDim + jr];

							}
						}
					}
			}
			else
			{
				int randomBot = fitnessIndex[0];
				for (int layerT = 0; layerT < Const->TopologySize - 1; layerT++)
				{
					for (int it = 0; it < Topology[layerT] + 1; it++)
					{
						for (int jt = 0; jt < Topology[layerT + 1]; jt++)
						{
							RNN->NNweights[((pos*Const->TopologySize + layerT)*ArrayDim + it)*ArrayDim + jt] =
								RNN->NNweights[((randomBot*Const->TopologySize + layerT)*ArrayDim + it)*ArrayDim + jt];

						}
					}
				}

				for (int layerR = 0; layerR < Const->TopologyRNNSize - 1; layerR++)
				{
					for (int ir = 0; ir < TopologyRNN[layerR] + 1; ir++)
					{
						for (int jr = 0; jr < TopologyRNN[layerR]; jr++)
						{
							RNN->MNweights[((pos*Const->TopologySize + layerR)*ArrayDim + ir)*ArrayDim + jr] =
								RNN->MNweights[((randomBot*Const->TopologySize + layerR)*ArrayDim + ir)*ArrayDim + jr];

						}
					}
				}

			}

		}
		else
		{

			if (pos != fitnessIndex[0])
			{
				float crossValue = crossValueBig;

				if (bot[pos].Fitness > fitness5)
				{
					crossValue = crossValueSmall;
				}


				float minValue = 0.65f*numElements;
				float maxValue = numElements-1;

				float range = maxValue - minValue;
				int randomBotIndex = (int)((fabs(rnnDNA[pos*numElements +1])* range + minValue));
				int randomBotIndex2 = (int)((fabs(rnnDNA[pos*numElements + 2]) * range + minValue));
				int randomBot = fitnessIndex[randomBotIndex];
				int randomBot2 = fitnessIndex[randomBotIndex2];

				int rndstep = 0;
				for (int layerT = 0; layerT < Const->TopologySize - 1; layerT++)
				{
					for (int it = 0; it < Topology[layerT] + 1; it++)
					{
						for (int jt = 0; jt < Topology[layerT + 1]; jt++)
						{
							RNN->NNweights[((pos*Const->TopologySize + layerT)*ArrayDim + it)*ArrayDim + jt] =
								RNN->NNweights[((randomBot*Const->TopologySize + layerT)*ArrayDim + it)*ArrayDim + jt];
							if (fabs(rnnDNA[pos*numElements + 3+ rndstep])<crossValue)
								RNN->NNweights[((pos*Const->TopologySize + layerT)*ArrayDim + it)*ArrayDim + jt] =
								RNN->NNweights[((randomBot2*Const->TopologySize + layerT)*ArrayDim + it)*ArrayDim + jt];
							rndstep++;
						}
					}
				}
				 rndstep = 0;
				for (int layerR = 0; layerR < Const->TopologyRNNSize - 1; layerR++)
				{
					for (int ir = 0; ir < TopologyRNN[layerR] + 1; ir++)
					{
						for (int jr = 0; jr < TopologyRNN[layerR]; jr++)
						{
							RNN->MNweights[((pos*Const->TopologySize + layerR)*ArrayDim + ir)*ArrayDim + jr] =
								RNN->MNweights[((randomBot*Const->TopologySize + layerR)*ArrayDim + ir)*ArrayDim + jr];
							if (fabs(rnnDNA[pos*numElements + 25 + rndstep])<crossValue)
								RNN->MNweights[((pos*Const->TopologySize + layerR)*ArrayDim + ir)*ArrayDim + jr] =
								RNN->MNweights[((randomBot2*Const->TopologySize + layerR)*ArrayDim + ir)*ArrayDim + jr];
							rndstep++;

						}
					}
				}

//mutation
			rndstep = 0;
				for (int layerT = 0; layerT < Const->TopologySize - 1; layerT++)
				{
					for (int it = 0; it < Topology[layerT] + 1; it++)
					{
						for (int jt = 0; jt < Topology[layerT + 1]; jt++)
						{
							
							if (fabs(rnnDNA[pos*numElements + 50 + rndstep])< 0.0015)
							{
								if (fabs(rnnDNA[pos*numElements + 75 + rndstep]) < 0.5)
									RNN->NNweights[((pos*Const->TopologySize + layerT)*ArrayDim + it)*ArrayDim + jt] =
									-RNN->NNweights[((pos*Const->TopologySize + layerT)*ArrayDim + it)*ArrayDim + jt]/30.f;
								else
									RNN->NNweights[((pos*Const->TopologySize + layerT)*ArrayDim + it)*ArrayDim + jt] =
									+RNN->NNweights[((pos*Const->TopologySize + layerT)*ArrayDim + it)*ArrayDim + jt]/30.f;
								rndstep++;
							}
						}
					}
				}
				rndstep = 0;
				for (int layerR = 0; layerR < Const->TopologyRNNSize - 1; layerR++)
				{
					for (int ir = 0; ir < TopologyRNN[layerR] + 1; ir++)
					{
						for (int jr = 0; jr < TopologyRNN[layerR]; jr++)
						{
							
							if (fabs(rnnDNA[pos*numElements + 100 + rndstep])< 0.0015)
							{
								if (fabs(rnnDNA[pos*numElements + 125 + rndstep]) < 0.5)
									RNN->MNweights[((pos*Const->TopologySize + layerR)*ArrayDim + ir)*ArrayDim + jr] =
									-RNN->MNweights[((pos*Const->TopologySize + layerR)*ArrayDim + ir)*ArrayDim + jr] / 30.f;
								else
									RNN->MNweights[((pos*Const->TopologySize + layerR)*ArrayDim + ir)*ArrayDim + jr] =
									+RNN->MNweights[((pos*Const->TopologySize + layerR)*ArrayDim + ir)*ArrayDim + jr] / 30.f;
								rndstep++;
							}

						}
					}
				}
			}

		}
		
	}
}


__global__ void cudaBeforeNNRNN(Bot *bot, float *gbotSector, float *tgbotSector, argumentsRNN *RNN, ConstantStruct *ct, int numElements, int gameTick)
{
	const int     tid = blockIdx.x * blockDim.x + threadIdx.x;
	const int     threadN = gridDim.x * blockDim.x;

	const int numSensorElem = 2 * ct->numberofSensorshalf;

	for (int pos = tid; pos < numElements; pos += threadN)
	{
		const  int aa = 16 * pos;
		const  int ii = pos;
		const int iiA = ii*ct->ArrayDim;

		if(gameTick!=0)
		{ 
			for (int r = 0; r < numSensorElem; r++)
			{
				RNN->outputs[iiA + r] = gbotSector[numSensorElem*ii + r];
				
			}
			for (int r = 0; r < numSensorElem; r++)
			{
				RNN->outputs[iiA + r+ numSensorElem] = tgbotSector[numSensorElem*ii + r];
				tgbotSector[numSensorElem*ii + r]= gbotSector[numSensorElem*ii + r];
			}
			RNN->outputs[iiA + 2*numSensorElem] = bot[pos].Mass/80.f;
		    RNN->outputs[iiA + 2*numSensorElem+1]= bot[pos].speed/ 35.f;
			RNN->outputs[iiA + 2*numSensorElem + 2] = bot[pos].angle / 3.14f;
			
		}
		else
		{
			for (int r = 0; r < numSensorElem; r++)
			{
				RNN->outputs[iiA + r] = gbotSector[numSensorElem*ii + r];
			}
			for (int r = numSensorElem; r < 2 * numSensorElem; r++)
			{
				RNN->outputs[iiA + r] = 0;
			}
			RNN->outputs[iiA + numSensorElem] = bot[pos].Mass / 80.f;
			RNN->outputs[iiA + numSensorElem + 1] = bot[pos].speed / 35.f;
			RNN->outputs[iiA + numSensorElem + 2] = bot[pos].angle / 3.14f;
			RNN->outputs[iiA + numSensorElem + 3] = (bot[pos].X - ct->GAME_WIDTH / 2) / ct->GAME_WIDTH / 2;
			RNN->outputs[iiA + numSensorElem + 4] = (bot[pos].Y- ct->GAME_HEIGHT / 2) / ct->GAME_HEIGHT / 2;


		}
	}

}




__global__ void cudaServer(Bot *bot, Food *food,float *gbotSector, ConstantStruct *ct, int numElements, int gameTick)
{
	int  tid = blockIdx.x * blockDim.x + threadIdx.x;
	int  threadN = gridDim.x * blockDim.x;


	int gWidth = ct->GAME_WIDTH;
	int gHeight = ct->GAME_HEIGHT;

	const int ListSize = ct->DepthBotPopulation;
	//const int VirusListSize = ct->VirusPopulationSize;
	const int FoodListSize = ct->DepthFoodPopulation;

	const int conworldangleStepInit = ct->angleStepInit;
	const int AngleStep = ct->angleStep;
	const int numberofSensorshalf = ct->numberofSensorshalf;
	const int numSensorElem = 2 * numberofSensorshalf;

	const float speedfactor = ct->SPEED_FACTOR;
	const float inertionfactor = ct->INERTION_FACTOR;

	const float RadiusView = ct->RadiusOfView;

	const  int MaxDepth = ct->Depth;

	const float PI = 3.141592653f;

	const float RadtoGrad = (180.f / PI);

	
	for (int i= tid; i < numElements; i += threadN)
	{

		const int ii =i;


		if (bot[i].Mass > 0.f && bot[i].TTF != 0)
		{

			bot[i].Lifetime = bot[i].Lifetime + 1.f;

			bot[i].Radius= 2.f* sqrt(bot[i].Mass);

			if ((bot[i].X - bot[i].Radius) < 0.f)
			{
				bot[i].X = bot[i].Radius;
				bot[i].Xvelocity = 0.f;
			}

			if ((bot[i].X + bot[i].Radius)> gWidth)
			{
				bot[i].X = bot[i].X - bot[i].Radius+ bot[i].X - gWidth;
				bot[i].Xvelocity = 0.f;
			}

			if (bot[i].Y - bot[i].Radius< 0.f)
			{
				bot[i].Y = bot[i].Y + bot[i].Radius- bot[i].Y;
				bot[i].Yvelocity = 0.f;
			}
			if (bot[i].Y + bot[i].Radius> gHeight)
			{
				bot[i].Y = bot[i].Y - bot[i].Radius+ bot[i].Y - gHeight;
				bot[i].Yvelocity = 0.f;
			}

			float speed_x = bot[i].speed * cosf(bot[i].angle);
			float speed_y = bot[i].speed * sinf(bot[i].angle);

			float maxspeed = speedfactor / sqrtf(bot[i].Mass);

			float dx1 = bot[i].xPOI - bot[i].X;
			float dy1 = bot[i].yPOI - bot[i].Y;

			float distpoi = sqrt(dx1 * dx1 + dy1 * dy1);

			float nx = (distpoi > 0.f) ? (dx1 / distpoi) : 0.f;
			float ny = (distpoi > 0.f) ? (dy1 / distpoi) : 0.f;

			speed_x += (nx * maxspeed - speed_x) * inertionfactor / bot[i].Mass;
			speed_y += (ny * maxspeed - speed_y) * inertionfactor / bot[i].Mass;

			bot[i].angle = atan2f(speed_y, speed_x);

			float new_speed = sqrtf(speed_x * speed_x + speed_y * speed_y);

			if (new_speed > maxspeed)
			{
				new_speed = maxspeed;
			}

			bot[i].speed = new_speed;

			float x = bot[i].X;
			float y = bot[i].Y;
			float radius = bot[i].Radius;
			float speed = bot[i].speed;

			float rB = x + radius;
			float lB = x - radius;
			float dB = y + radius;
			float uB = y - radius;

			float dx = speed * cosf(bot[i].angle);
			float dy = speed * sinf(bot[i].angle);

			if ((rB + dx) < gWidth && (lB + dx) > 0.f)
			{
				bot[i].X = bot[i].X + dx;
			}
			else
			{
				// зануляем проекцию скорости по dx
				speed_y = speed * sinf(bot[i].angle);
				bot[i].speed = fabs(speed_y);
				bot[i].angle = (speed_y >= 0.f) ? PI / 2.f : -PI / 2.f;
			}

			if ((dB + dy) < gHeight && (uB + dy) > 0.f)
			{
				bot[i].Y = bot[i].Y + dy;
			}
			else
			{
				// зануляем проекцию скорости по dy
				speed_x = speed * cosf(bot[i].angle);
				bot[i].speed = fabs(speed_x);
				bot[i].angle = (speed_x >= 0.f) ? 0.f : PI;
			}

			bot[i].Xvelocity = bot[i].speed * cosf(bot[i].angle);
			bot[i].Yvelocity = bot[i].speed * sinf(bot[i].angle);

			
			if (bot[i].X < 1.5f * bot[i].Radius)
			{
				bot[i].Mass = bot[i].Mass - 0.25f;
				// bot[i].Fitness = bot[i].Fitness - 0.15; // Fitness
			}
			if (bot[i].X > gWidth - 1.5f* bot[i].Radius)
			{
				bot[i].Mass = bot[i].Mass - 0.25f;
				// bot[i].Fitness = bot[i].Fitness - 0.15; // Fitness
			}
			if (bot[i].Y < 1.5f * bot[i].Radius)
			{
				bot[i].Mass = bot[i].Mass - 0.25f;
				// bot[i].Fitness = bot[i].Fitness - 0.15; // Fitness
			}
			if (bot[i].Y > gHeight - 1.5f* bot[i].Radius)
			{
				bot[i].Mass = bot[i].Mass - 0.25f;
				// bot[i].Fitness = bot[i].Fitness - 0.15; // Fitness
			}

			//Bot

			int ibb = ct->BotPopulationSize *bot[i].Depth;
			for (int bb = ibb; bb < ibb + ct->BotPopulationSize; ++bb)
				//for (int bbc = 0; bbc < ListSize; bbc++)
			{
			
				if (bot[i].Depth == bot[bb].Depth && i != bb
					&& bot[bb].Mass > 0.f && (int)bot[bb].TTF != 0 && fabs(bot[i].X - bot[bb].X) +
					fabs(bot[i].Y - bot[bb].Y)
					<= 2.f *RadiusView* bot[i].Radius)
				{

					float dxa = bot[i].X - bot[bb].X;
					float dya = bot[i].Y - bot[bb].Y;

					float distance = sqrtf(dxa * dxa + dya * dya);

					if (distance < bot[i].Radius- 1.f * bot[bb].Radius / 3.f &&
						bot[i].Mass > 1.2f * bot[bb].Mass )
					{
						bot[i].Mass = bot[i].Mass + bot[bb].Mass/3 ;

						bot[bb].Mass = 0.f;
						bot[bb].TTF = 0.f;

						bot[i].Fitness = bot[i].Fitness + 55.f; // Fitness

					}

				}

			}


			//food

			//for (int ffc = 0; ffc < FoodListSize; ffc++)

			int iff = 4 * ct->FoodPopulationSize *bot[i].Depth;
			for (int ff = iff; ff < iff + 4 * ct->FoodPopulationSize; ff++)
			{
				if (bot[i].Depth == food[ff].Depth &&
					food[ff].Mass > 0.f && fabs(bot[i].X - food[ff].X) +
					fabs(bot[i].Y - food[ff].Y)
					<= 2.f *RadiusView* bot[i].Radius)
				{

					float dxf = bot[i].X - food[ff].X;
					float dyf = bot[i].Y - food[ff].Y;

					float distance = sqrtf(dxf * dxf + dyf * dyf);

					if (distance < bot[i].Radius+ food[ff].Radius)
					{
						bot[i].Mass = bot[i].Mass + food[ff].Mass;
						food[ff].Mass = -1.f;
						bot[i].Fitness = bot[i].Fitness + 3.5f;
					}
				}

			}

			bot[i].Mass = bot[i].Mass - 0.035f;

			if (bot[i].Mass <= 15.f)
			{
				bot[i].Fitness = bot[i].Fitness ;
				bot[i].TTF = 0.f;
			}
		}

		if (bot[i].Mass > 0.f && bot[i].TTF != 0)
		{

			for (int isec = 0; isec < 2 * numberofSensorshalf; isec++)
			{
				gbotSector[numSensorElem*ii + isec] = 0.f;
			}


			float deltaspeedx = bot[i].Xvelocity;
			float deltaspeedy = bot[i].Yvelocity;

			float distspeed = sqrtf(deltaspeedx * deltaspeedx + deltaspeedy * deltaspeedy);

			bot[i].nXvelocity = 0.f;
			bot[i].nYvelocity = 0.f;

			if (distspeed > 0.f)
			{
				bot[i].nXvelocity = bot[i].Xvelocity / distspeed;
				bot[i].nYvelocity = bot[i].Yvelocity / distspeed;
			}


			bool wallflag = false;

			if (fabs(bot[i].Xvelocity) < 0.0001f)
				bot[i].Xvelocity = 0.f;
			if (fabs(bot[i].Yvelocity) < 0.0001f)
				bot[i].Yvelocity = 0.f;

			float DistanceToPOI = 0.f;

			int SensorInputNumber1 = 0;

			for (int tw = conworldangleStepInit; tw < 180; tw = tw + AngleStep)
			{
				float ThetaDegree1 = tw;
				float ThetaRad1 = ThetaDegree1 * PI / 180.f;
				float cs1 = cosf(ThetaRad1);
				float sn1 = sinf(ThetaRad1);

				float NormVectoX = 0.f;
				float NormVEctorY = 1.f;

				if (bot[i].nXvelocity != 0.f && bot[i].nYvelocity != 0.f)
				{
					NormVectoX =  bot[i].nXvelocity;
					NormVEctorY = bot[i].nYvelocity;
				}

				float VEctorXrotate = bot[i].X + RadiusView * bot[i].Radius*
					(cs1 * NormVectoX - sn1 * NormVEctorY);
				float VEctorYrotate = bot[i].Y + RadiusView * bot[i].Radius*
					(sn1 * NormVectoX + cs1 * NormVEctorY);


				float ThetaDegree2 = tw + AngleStep;
				float ThetaRad2 = ThetaDegree2 * PI / 180.f;
				float cs2 = cosf(ThetaRad2);
				float sn2 = sinf(ThetaRad2);


				float VEctorXrotate2 = bot[i].X + RadiusView * bot[i].Radius*
					(cs2 * NormVectoX - sn2 * NormVEctorY);
				float VEctorYrotate2 = bot[i].Y + RadiusView * bot[i].Radius*
					(sn2 * NormVectoX + cs2 * NormVEctorY);


				if ((VEctorXrotate > gWidth || VEctorXrotate < 0.f) && (VEctorXrotate2 > gWidth || VEctorXrotate2 < 0.f)
					|| (VEctorYrotate > gHeight || VEctorYrotate < 0.f) && (VEctorYrotate2 > gHeight || VEctorYrotate2 < 0.f)
					)
				{

					gbotSector[numSensorElem*ii + SensorInputNumber1] = -0.75f;
					wallflag = true;
				}

				VEctorXrotate = bot[i].X + RadiusView * bot[i].Radius*
					(cs1 * NormVectoX + sn1 * NormVEctorY);
				VEctorYrotate = bot[i].Y + RadiusView * bot[i].Radius*
					(-sn1 * NormVectoX + cs1 * NormVEctorY);


				VEctorXrotate2 = bot[i].X + RadiusView * bot[i].Radius*
					(cs2 * NormVectoX + sn2 * NormVEctorY);
				VEctorYrotate2 = bot[i].Y + RadiusView * bot[i].Radius*
					(-sn2 * NormVectoX + cs2 * NormVEctorY);


				if ((VEctorXrotate > gWidth || VEctorXrotate < 0.f) && (VEctorXrotate2 > gWidth || VEctorXrotate2 < 0.f)
					|| (VEctorYrotate > gHeight || VEctorYrotate < 0.f) && (VEctorYrotate2 > gHeight || VEctorYrotate2 < 0.f)
					)
				{

					gbotSector[numSensorElem*ii + SensorInputNumber1 + numberofSensorshalf] = -0.75f;
					wallflag = true;
				}

				SensorInputNumber1++;

			}



			//Food Collisions
			//  if (wallflag == false)

			//for (int ffc = 0; ffc < FoodListSize; ffc++)

			//food
			int iff = 4 *ct->FoodPopulationSize *bot[i].Depth;
			for (int ff = iff; ff < iff + 4 * ct->FoodPopulationSize; ff++)
			{
				if (food[ff].Mass>0.f && bot[i].Depth == food[ff].Depth &&
					fabs(bot[i].X - food[ff].X) +
					fabs(bot[i].Y - food[ff].Y)
					<= 2.f *RadiusView* bot[i].Radius)
				{

					float dxa = food[ff].X - bot[i].X;
					float dya = food[ff].Y - bot[i].Y;

					float distance_to_unita = sqrtf(dxa * dxa + dya * dya);
					float nXalien = 0.f;
					float nYalien = 0.f;

					if (distance_to_unita > 0.)
					{
						nXalien = dxa / distance_to_unita;
						nYalien = dya / distance_to_unita;


						if ((distance_to_unita - (food[ff].Radius + 0.03f)) <= RadiusView * bot[i].Radius)
						{

							int SensorInputNumber = 0;

							for (int t = conworldangleStepInit; t < 180; t = t + AngleStep)
							{

								float angleAttack =
									anglebetween2vectors(bot[i].Xvelocity, bot[i].Yvelocity, nXalien, nYalien) * RadtoGrad;



								if (fabs(angleAttack) > t &&
									fabs(angleAttack) <= t + AngleStep)
								{
									if (angleAttack > 0.f)
									{
										gbotSector[numSensorElem*ii + SensorInputNumber] =
											gbotSector[numSensorElem*ii + SensorInputNumber] +
											(RadiusView * bot[i].Radius- distance_to_unita) / (RadiusView * bot[i].Radius* 2.f);



									}
									else
									{
										gbotSector[numSensorElem*ii + SensorInputNumber + numberofSensorshalf] =
											gbotSector[numSensorElem*ii + SensorInputNumber + numberofSensorshalf] +
											(RadiusView * bot[i].Radius- distance_to_unita) / (RadiusView * bot[i].Radius* 2.f);

									}
								}
								SensorInputNumber++;

							}

						}
					}
				}
			}

			//Alien Collisions
			//  if (wallflag==false)
			//for (int bbc = 0; bbc < ListSize; bbc++)

			int ibb = ct->BotPopulationSize *(int)bot[i].Depth;
			for (int bb = ibb; bb < ibb + ct->BotPopulationSize; ++bb)
			{

				if (bot[i].Depth == bot[bb].Depth && i != bb
					&& bot[bb].Mass > 0.f && (int)bot[bb].TTF != 0 && fabs(bot[i].X - bot[bb].X) +
					fabs(bot[i].Y - bot[bb].Y)
					<= 2.f *RadiusView* bot[i].Radius)
				{

					float dxa = bot[bb].X - bot[i].X;
					float dya = bot[bb].Y - bot[i].Y;

					float distance_to_unita = sqrtf(dxa * dxa + dya * dya);

					float nXalien = 0.f;
					float nYalien = 0.f;

					if (distance_to_unita > 0)
					{
						nXalien = dxa / distance_to_unita;
						nYalien = dya / distance_to_unita;


						if (distance_to_unita - bot[bb].Radius <= RadiusView * bot[i].Radius)
						{

							int SensorInputNumber = 0;

							for (int t = conworldangleStepInit; t < 180; t = t + AngleStep)
							{


								float angleAttack =
									anglebetween2vectors(bot[i].Xvelocity, bot[i].Yvelocity, nXalien, nYalien) * RadtoGrad;


								if (fabs(angleAttack) > t &&
									fabs(angleAttack) <= t + AngleStep)
								{
									if (angleAttack > 0.f)
									{

										if (bot[i].Mass > 1.2f * bot[bb].Mass)
										{
											gbotSector[numSensorElem*ii + SensorInputNumber] =
												gbotSector[numSensorElem*ii + SensorInputNumber] +
												0.95f * (RadiusView * bot[i].Radius- distance_to_unita) / (RadiusView* bot[i].Radius);

										}
										if (bot[i].Mass <= 1.2f * bot[bb].Mass)
										{
											gbotSector[numSensorElem*ii + SensorInputNumber] =
												-0.95f * (RadiusView * bot[i].Radius- distance_to_unita) / (RadiusView* bot[i].Radius);

										}

									}
									else
									{
										if (bot[i].Mass > 1.2f * bot[bb].Mass)
										{
											gbotSector[numSensorElem*ii + SensorInputNumber + numberofSensorshalf] =
												gbotSector[numSensorElem*ii + SensorInputNumber + numberofSensorshalf] +
												0.95f * (RadiusView * bot[i].Radius- distance_to_unita) / (RadiusView * bot[i].Radius);

										}
										if (bot[i].Mass <= 1.2f * bot[bb].Mass)
										{
											gbotSector[numSensorElem*ii + SensorInputNumber + numberofSensorshalf] =
												-0.95f * (RadiusView * bot[i].Radius- distance_to_unita) / (RadiusView * bot[i].Radius);

										}


									}
								}
								SensorInputNumber++;

							}
						}

					}

				}
			}


		}

	}
}

__global__ void cudaAfterNNRNN(Bot *bot, float *gbotSector, argumentsRNN *RNN, ConstantStruct *ct, int numElements, int gameTick,
	int *fitnessIndex, int* fitnessValue)
{
	int  tid = blockIdx.x * blockDim.x + threadIdx.x;
	int  threadN = gridDim.x * blockDim.x;

	int numberLayers =ct->TopologySize;
	const float PI = 3.141592653f;
	const float RadtoGrad = (180.f / PI);

	int LayerDim = ct->ArrayDim;

	for (int i = tid; i< numElements; i += threadN)
	{

		fitnessValue[i] = bot[i].Fitness;
		fitnessIndex[i] = bot[i].intID;
		
		if (bot[i].Mass > 0.f && bot[i].TTF != 0)
		{
			const int iiA = i*ct->ArrayDim;

			float rotate1 = RNN->outputs[iiA + 0];
			float rotate2 = RNN->outputs[iiA + 1];
			float speedFoodValue1 = RNN->outputs[iiA + 2];
			float speedFoodValue2 = RNN->outputs[iiA + 3];


			//float rotateAlertValue = layer.Neurons[2].Value;
			// float speedAlertValue =  layer.Neurons[3].Value;

			float angle = bot[i].angle;
			float speed = bot[i].speed;

			float criteria = 0.f;

			if (rotate1 > criteria && rotate2 < criteria)
			{
				angle = angle + 12.f * PI / 180.f;

			}
			if (rotate1 < criteria && rotate2 > criteria)
			{
				angle = angle - 12.f * PI / 180.f;

			}

			if (speedFoodValue1 > criteria && speedFoodValue2 <criteria)
			{
				speed = speed + 2.7f;
			}
			if (speedFoodValue1 < criteria && speedFoodValue2 > criteria)
			{
				speed = speed - 2.7f;
			}


			float dx = speed * cos(angle);
			float dy = speed * sin(angle);

			bot[i].xPOI = (bot[i].X+ dx);
			bot[i].yPOI = (bot[i].Y+ dy);


			for (int i2 = 0; i2 < ct->ArrayDim; i2++)
			{
				RNN->outputs[iiA + i2] = 0.f;
			}


		}



	}
}

__global__ void smallinitNNRNN(Bot *bot, argumentsRNN *RNN, ConstantStruct *Const, int numElements, int gameTick)
{
	int  tid = blockIdx.x * blockDim.x + threadIdx.x;
	int  threadN = gridDim.x * blockDim.x;

	

	for (int pos = tid; pos < numElements; pos += threadN)
	{
		const  int ii = pos;
		const int iiA = pos*Const->ArrayDim;
		int LayerDim = Const->ArrayDim;

		for (int i2 = 0; i2 < LayerDim; i2++)
		{
			RNN->outputs[iiA + i2] = 0.f;
		}

	}
}

__global__ void initNNRNN(Bot *bot, argumentsRNN *RNN, ConstantStruct *Const, int numElements, int gameTick)
{
	int  tid = blockIdx.x * blockDim.x + threadIdx.x;
	int  threadN = gridDim.x * blockDim.x;



	for (int pos = tid; pos < numElements; pos += threadN)
	{
		const  int ii = pos;
		const int iiA = pos*Const->ArrayDim;
		int LayerDim = Const->ArrayDim;

		for (int i2 = 0; i2 < LayerDim; i2++)
		{
			RNN->outputs[iiA + i2] = 0.f;
		}

		bot[pos].Fitness = 100000;

	}
}

__global__ void initkernelRNN(float *outputs, float *sums, float *sumsContext, float *NNweights, float *MNweights,
	  float *neuronContext,  argumentsRNN *out)
{
	if (threadIdx.x == 0) {

		out->outputs = outputs;
		out->sums = sums;
		out->sumsContext = sumsContext;
		out->NNweights = NNweights;
		out->MNweights = MNweights;
		out->neuronContext = neuronContext;
	}
}

__global__ void cudaRNN(Bot *bot, argumentsRNN *RNN, ConstantStruct *Const, int *Topology, int *TopologyRNN, int numElements, int gameTick)
{
	int  tid = blockIdx.x * blockDim.x + threadIdx.x;
	int  threadN = gridDim.x * blockDim.x;

	int TopologySize = Const->TopologySize;
	

	for (int pos = tid; pos < numElements; pos += threadN)
	{
		const  int ii = pos;
		const int iiA = pos*Const->ArrayDim;
		int ArrayDim = Const->ArrayDim;
		const int iiAT = ii*TopologySize*ArrayDim;

		if (bot[pos].TTF != 0 && bot[pos].Mass>0)
		{
			//RNN->outputs[iiA + Topology[0]] = RNN->NNweights[((ii*TopologySize + layer1)*ArrayDim + i5)*ArrayDim + j4]; //1.f;
			//bias neurons[iiA +  Topology[0]] = 1;

			int neuroncount7 = Topology[0];

			neuroncount7++;

			for (int layer1 = 0; layer1 < TopologySize - 1; layer1++)
			{
				RNN->outputs[iiA + Topology[0]] = RNN->NNweights[((ii*TopologySize + layer1)*ArrayDim + ArrayDim-1)*ArrayDim + ArrayDim-1]; //1.f;
				for (int j4 = 0; j4 < Topology[layer1 + 1]; j4++)
				{
					for (int i5 = 0; i5 < Topology[layer1] + 1; i5++)
					{
						RNN->sums[iiA + j4] = RNN->sums[iiA + j4] + RNN->outputs[iiA + i5] *
							RNN->NNweights[((ii*TopologySize + layer1)*ArrayDim + i5)*ArrayDim + j4];
					}
				}

				if (TopologyRNN[layer1] > 0)
				{

					for (int j14 = 0; j14 < Topology[layer1]; j14++)
					{
						for (int i15 = 0; i15 < Topology[layer1]; i15++)
						{


							RNN->sumsContext[iiA + j14] = RNN->sumsContext[iiA + j14] +
								RNN->neuronContext[iiAT + ArrayDim * layer1 + i15] *
								RNN->MNweights[((ii*TopologySize + layer1)*ArrayDim + i15)*ArrayDim + j14];

						}
						RNN->sumsContext[iiA + j14] = RNN->sumsContext[iiA + j14] + RNN->MNweights[((ii*TopologySize + layer1)*ArrayDim + ArrayDim - 1)*ArrayDim + ArrayDim - 1]*//1.0f*
							RNN->MNweights[((ii*TopologySize + layer1)*ArrayDim + Topology[layer1])*ArrayDim + j14]; //bias=1
					}

					for (int t = 0; t < Topology[layer1 + 1]; t++)
					{

						RNN->outputs[iiA + t] = Tanh(RNN->sums[iiA + t] + RNN->sumsContext[iiA + t]);
						RNN->neuronContext[iiAT + ArrayDim * layer1 + t] = RNN->outputs[iiA + t];

						//neurons[iiA + neuroncount7] = outputs[iiA + t];
						neuroncount7++;
					}
					//SoftMax
/*
					double sum = 0.0;
					for (int k = 0; k <ArrayDim; ++k)
						sum += exp(RNN->outputs[iiA + k]);

					for (int k = 0; k < ArrayDim; ++k)
						RNN->outputs[iiA + k] = exp(RNN->outputs[iiA + k]) / sum;
*/

				}
				else
				{
					for (int t= 0; t < Topology[layer1 + 1]; t++)
					{

						RNN->outputs[iiA + t] = Tanh(RNN->sums[iiA + t]); //sigma

																			 //neurons[iiA + neuroncount7] = outputs[iiA + i1];
						neuroncount7++;
					}

				}


				if (layer1 + 1 != TopologySize - 1)
				{
					RNN->outputs[iiA + Topology[layer1 + 1]] = RNN->NNweights[((ii*TopologySize + layer1+1)*ArrayDim + ArrayDim - 1)*ArrayDim + ArrayDim - 1];// 1.f;
					//neurons[iiA + neuroncount7] = 1;
					neuroncount7++;
				}


				for (int i2 = 0; i2 < ArrayDim; i2++)
				{
					RNN->sums[iiA + i2] = 0.f;
					RNN->sumsContext[iiA + i2] = 0.f;

				}

			}

			
		}

	}
}


__global__ void cudaRnd(float *RndDNA, int numElements)
{
	int  tid = blockIdx.x * blockDim.x + threadIdx.x;
	int  threadN = gridDim.x * blockDim.x;

	//float w = sqrt(3.0f / 2.f);

	for (int pos = tid; pos < numElements; pos += threadN)
	{
		
		RndDNA[pos] = 1.9999f * RndDNA[pos] - 0.9999f;
		//RndDNA[pos] = 2.0f *w* RndDNA[pos] - 1.0f*w;
	}
}

__host__ void CalculateSizeRNN(std::vector<int>& Topology, std::vector<int>& TopologyRNN, int& neuroncount,
	int& dendritecount)
{
	
	for (int i : Topology)
		neuroncount += i;
	for (int i : TopologyRNN)
		neuroncount += i;

	for (int layer1 = 0; layer1 <Topology.size() - 1; layer1++)
	{
		for (int i = 0; i < Topology[layer1] + 1; i++)
			for (int j = 0; j < Topology[layer1 + 1]; j++)
				dendritecount++;
	}


	for (int layer2 = 0; layer2 < TopologyRNN.size() - 1; layer2++)
	{
		for (int i7 = 0; i7 <TopologyRNN[layer2] + 1; i7++)
			for (int j7 = 0; j7 < TopologyRNN[layer2]; j7++)
				dendritecount++;
	}

}


void ComputeLSTM(Constant&  constant)
{

	size_t sizeTopology = constant.Topology.size() * sizeof(int);
	size_t sizeTopologyRNN = constant.TopologyRNN.size() * sizeof(int);
	int    *h_Topology = (int *)malloc(sizeTopology);
	int    *h_TopologyRNN = (int *)malloc(sizeTopologyRNN);
	for (int i = 0; i < constant.Topology.size(); ++i)
	{
		h_Topology[i] = constant.Topology[i];
		h_TopologyRNN[i] = constant.TopologyRNN[i];
	}
	int *d_Topology = NULL;
	hipMalloc((void **)&d_Topology, sizeTopology);
	int *d_TopologyRNN = NULL;
	hipMalloc((void **)&d_TopologyRNN, sizeTopologyRNN);
	hipMemcpy(d_Topology, h_Topology, sizeTopology, hipMemcpyHostToDevice);
	hipMemcpy(d_TopologyRNN, h_TopologyRNN, sizeTopologyRNN, hipMemcpyHostToDevice);


	int neuroncount = 0;
	int dendritecount = 0;

	int DepthBotPopulation = constant.DepthBotPopulation;

	CalculateSizeRNN(constant.Topology, constant.TopologyRNN, neuroncount, dendritecount);

	int RnnDNAsize = 2*DepthBotPopulation*constant.Topology.size()* constant.ArrayDim*constant.ArrayDim;
	int numberOfGensRNN = DepthBotPopulation*dendritecount;

	float *hrnnDNA;
	float *drnnDNA;
	size_t rnnDNASize = RnnDNAsize * sizeof(float);
	hrnnDNA = (float *)malloc(rnnDNASize);
	hipMalloc((void**)&drnnDNA, rnnDNASize);

	hiprandGenerator_t gen3r;
	//Set the generator options
	hiprandCreateGenerator(&gen3r, HIPRAND_RNG_PSEUDO_DEFAULT);
	//Generate random numbers
	hiprandSetPseudoRandomGeneratorSeed(gen3r, 1234ULL);
	hiprandGenerateUniform(gen3r, drnnDNA, RnnDNAsize);
	//hiprandDestroyGenerator(gen3r);

	cudaRnd <<< int(1 + RnnDNAsize / 32), 64 >> > (drnnDNA, RnnDNAsize);
	hipDeviceSynchronize();

	hipMemcpy(hrnnDNA, drnnDNA, rnnDNASize, hipMemcpyDeviceToHost);
	std::vector<float> rnnDNA(hrnnDNA, hrnnDNA + RnnDNAsize);


	//RNN coeff

	float *outputs, *sums, *sumsContext, *NNweights, *MNweights, *neuronContext;
	float *_outputs, *_sums, *_sumsContext, *_NNweights, *_MNweights, *_neuronContext;

	argumentsRNN *_argsRNN;
	
	int preSizeOutputs = DepthBotPopulation * constant.ArrayDim;
	
	int preSizeNweight = DepthBotPopulation*constant.Topology.size()  * constant.ArrayDim * constant.ArrayDim;
	int preSizeMweight = DepthBotPopulation*constant.TopologyRNN.size()  * constant.ArrayDim * constant.ArrayDim;
	int preSizeNeuronContext = DepthBotPopulation*constant.TopologyRNN.size()  * constant.ArrayDim * constant.ArrayDim;

	size_t sizeOutputs = preSizeOutputs * sizeof(float);
	size_t sizeNweight = preSizeNweight * sizeof(float);
	size_t sizeMweight = preSizeMweight * sizeof(float);
	size_t SizeNeuronContext = preSizeNeuronContext * sizeof(float);

	outputs = (float *)malloc(sizeOutputs);
	sums = (float *)malloc(sizeOutputs);
	sumsContext = (float *)malloc(sizeOutputs);

	NNweights = (float *)malloc(sizeNweight);
	MNweights = (float *)malloc(sizeMweight);
	neuronContext = (float *)malloc(SizeNeuronContext);

	hipMalloc((void**)&_outputs, sizeOutputs);
	hipMalloc((void**)&_sums, sizeOutputs);
	hipMalloc((void**)&_sumsContext, sizeOutputs);

	hipMalloc((void**)&_NNweights, sizeNweight);
	hipMalloc((void**)&_MNweights, sizeMweight);
	hipMalloc((void**)&_neuronContext, SizeNeuronContext);

	hipMalloc((void**)&_argsRNN, sizeof(argumentsRNN));


	ConstantStruct cudaConst;
	ConstantStruct *d_cudaConst;

	//h_cudaConst = (ConstantStruct *)malloc(sizeof(ConstantStruct));
	hipMalloc((void**)&d_cudaConst, sizeof(ConstantStruct));

	cudaConst.FOOD_MASS=constant.FOOD_MASS;
	cudaConst.GAME_HEIGHT=constant.GAME_HEIGHT;
	cudaConst.GAME_TICKS=constant.GAME_TICKS;
	cudaConst.GAME_WIDTH=constant.GAME_WIDTH;
	cudaConst.INERTION_FACTOR=constant.INERTION_FACTOR;
	cudaConst.MAX_FRAGS_CNT=constant.MAX_FRAGS_CNT;
	cudaConst.SPEED_FACTOR=constant.SPEED_FACTOR;
	cudaConst.TICKS_TIL_FUSION=constant.TICKS_TIL_FUSION;
	cudaConst.VIRUS_RADIUS=constant.VIRUS_RADIUS;
	cudaConst.VIRUS_SPLIT_MASS=constant.VIRUS_SPLIT_MASS;
	cudaConst.VISCOSITY = constant.VISCOSITY;
	cudaConst.RadiusOfView=constant.RadiusOfView;

	cudaConst.SUM_RESP_TIMEOUT=constant.SUM_RESP_TIMEOUT;      
	cudaConst.RESP_TIMEOUT=constant.RESP_TIMEOUT ;        

	cudaConst.Depth=constant.Depth;
	cudaConst.CrossMove=constant.CrossMove;

	cudaConst.BotPopulationSize=constant.BotPopulationSize;
	cudaConst.VirusPopulationSize=constant.VirusPopulationSize;
	cudaConst.FoodPopulationSize=constant.FoodPopulationSize ;
	cudaConst.FoodAddPopulationSize=constant.FoodAddPopulationSize;
	cudaConst.FoodPAdd=constant.FoodPAdd;
	cudaConst.GlobalTicks=constant.GlobalTicks;

	cudaConst.DepthBotPopulation=constant.DepthBotPopulation;
	cudaConst.DepthFoodPopulation = constant.DepthFoodPopulation;
	cudaConst.DepthAddFoodPopulation = constant.DepthAddFoodPopulation;

	cudaConst.radius=constant.radius;
	cudaConst.KoeffWall=constant.KoeffWall;
	cudaConst.KoefEnemyDanger=constant.KoefEnemyDanger;
	cudaConst.KoefEnemyEat=constant.KoefEnemyEat;
	cudaConst.KoefEat=constant.KoefEat;
	cudaConst.KoefVirus=constant.KoefVirus;
	cudaConst.Lkoef=constant.Lkoef;

	cudaConst.xSetRight=constant.xSetRight;
	cudaConst.xSetLeft=constant.xSetLeft;
	cudaConst.ySetTop=constant.ySetTop;
	cudaConst.ySetDown=constant.ySetDown;

	cudaConst.IdeaTick=constant.IdeaTick;
	cudaConst.IdeaShortTick=constant.IdeaShortTick;

	cudaConst.numberofSensors=constant.numberofSensors;
	cudaConst.numberofSensorshalf=constant.numberofSensorshalf;
	cudaConst.numberofSensorsAll=constant.numberofSensorsAll;
	cudaConst.angleStep=constant.angleStep;
	cudaConst.angleStepInit=constant.angleStepInit;
	cudaConst.WorldTick=constant.WorldTick;
	cudaConst.ArrayDim=constant.ArrayDim;
	cudaConst.TopologySize = constant.Topology.size();
	cudaConst.TopologyRNNSize = constant.TopologyRNN.size();

    hipMemcpy(d_cudaConst, &cudaConst, sizeof(ConstantStruct), hipMemcpyHostToDevice);

	//initialization of Bot

	int numElements = constant.DepthBotPopulation;;

	//thrust::host_vector<Bot> thBotList;
	//thrust::device_vector<Bot> tdBotList;

	std:: vector<Bot> BotList(numElements);
	Bot *hBotList;
	Bot *dBotList;
	size_t BotSize = numElements * sizeof(Bot);
	hBotList = (Bot *)malloc(BotSize);
	hipMalloc((void**)&dBotList, BotSize);

	std::vector<Bot> BotListSort(numElements);
	Bot *hBotListSort;
	Bot *dBotListSort;
	hBotListSort = (Bot *)malloc(BotSize);
	hipMalloc((void**)&dBotListSort, BotSize);


	std::vector<Food> FoodList(constant.DepthFoodPopulation);
	Food *hFoodList;
	Food *dFoodList;
	size_t FoodSize = FoodList.size() * sizeof(Food);
	hFoodList = (Food *)malloc(FoodSize);
	hipMalloc((void**)&dFoodList, FoodSize);

	std::vector<Food> AddFoodList(constant.DepthAddFoodPopulation);
	Food *hAddFoodList;
	Food *dAddFoodList;
	size_t AddFoodSize = AddFoodList.size() * sizeof(Food);
	hAddFoodList = (Food *)malloc(AddFoodSize);
	hipMalloc((void**)&dAddFoodList, AddFoodSize);

	
	int numSensorElem = 2 * (int)constant.numberofSensorshalf;
	int numSensorElements = (int)BotList.size()*numSensorElem;
	std::vector<float> Sector(numSensorElements);
	std::fill(Sector.begin(), Sector.begin() + numSensorElements, 0);
	size_t sizeSensorElem = numSensorElements * sizeof(float);



	float *h_gbotSector = (float *)malloc(sizeSensorElem);
	float *d_gbotSector = NULL;
	hipMalloc((void **)&d_gbotSector, sizeSensorElem);

	float *h_tgbotSector = (float *)malloc(sizeSensorElem);
	float *d_tgbotSector = NULL;
	hipMalloc((void **)&d_tgbotSector, sizeSensorElem);

	for (int i = 0; i <preSizeOutputs; i++)
	{
		outputs[i] = 0;
		sums[i] = 0;
		sumsContext[i] = 0;
	}

	InitBotList(BotList, Sector, constant);

	for (Bot botIndex : BotList)
	{
		for (int layerT = 0; layerT < constant.Topology.size() - 1; layerT++)
		{
			for (int it = 0; it < constant.Topology[layerT] + 1; it++)
			{
				for (int jt = 0; jt < constant.Topology[layerT + 1]; jt++)
				{
					NNweights[((botIndex.intID*constant.Topology.size() + layerT)*constant.ArrayDim + it)*constant.ArrayDim + jt] =
						rnnDNA[((botIndex.intID*constant.Topology.size() + layerT)*constant.ArrayDim + it)*constant.ArrayDim + jt];
				}
			}
		}

		for (int layerR = 0; layerR < constant.TopologyRNN.size() - 1; layerR++)
		{
			for (int ir = 0; ir < constant.TopologyRNN[layerR] + 1; ir++)
			{
				for (int jr = 0; jr < constant.TopologyRNN[layerR]; jr++)
				{
					MNweights[((botIndex.intID*constant.Topology.size() + layerR)*constant.ArrayDim + ir)*constant.ArrayDim + jr]
						= rnnDNA[preSizeMweight+((botIndex.intID*constant.Topology.size() + layerR)*constant.ArrayDim + ir)*constant.ArrayDim + jr];
				}
			}
		}
	}


	for (int i = 0; i <preSizeNeuronContext; i++)
	{
		neuronContext[i] = 0;
	}

	hipMemcpy(_outputs, outputs, sizeOutputs, hipMemcpyHostToDevice);
	hipMemcpy(_sums, sums, sizeOutputs, hipMemcpyHostToDevice);
	hipMemcpy(_sumsContext, sumsContext, sizeOutputs, hipMemcpyHostToDevice);

	hipMemcpy(_NNweights, NNweights, sizeNweight, hipMemcpyHostToDevice);
	hipMemcpy(_MNweights, MNweights, sizeNweight, hipMemcpyHostToDevice);
	hipMemcpy(_neuronContext, neuronContext, SizeNeuronContext, hipMemcpyHostToDevice);

	initkernelRNN <<< 1, 1 >> > (_outputs, _sums, _sumsContext, _NNweights, _MNweights, _neuronContext, _argsRNN);
	hipDeviceSynchronize();


	std::vector<int> fitnessIndex;
	fitnessIndex.resize(DepthBotPopulation);
	std::vector<int> fitnessValue;
	fitnessValue.resize(DepthBotPopulation);

	int numFitnessElem = (int)DepthBotPopulation;
	size_t sizeFitness = numFitnessElem * sizeof(int);

	int *h_fitnessIndex = (int*)malloc(sizeFitness);
	int *h_fitnessValue = (int*)malloc(sizeFitness);

	int *d_fitnessIndex = NULL;
	hipMalloc((void **)&d_fitnessIndex, sizeFitness);
	int *d_fitnessValue = NULL;
	hipMalloc((void **)&d_fitnessValue, sizeFitness);

	hipMemcpy(d_fitnessIndex, h_fitnessIndex, sizeFitness, hipMemcpyHostToDevice);
	hipMemcpy(d_fitnessValue, h_fitnessValue, sizeFitness, hipMemcpyHostToDevice);



	//Init game loop

	GraphModule graphModule;
	graphModule.Init(constant.GAME_WIDTH, constant.GAME_HEIGHT, constant.Depth, constant.GAME_TICKS);

	InfoGraph infoGraph;
	infoGraph.Init();

	std::vector<float> rnnNNweights1(NNweights, NNweights + preSizeNweight);
	std::vector<float> rnnMNweights1(MNweights, MNweights + preSizeMweight);

	GraphGA graphGA;
	graphGA.Init(1700, 1250, 1700, 1250, constant.Depth);
	graphGA.Clear();
	graphGA.GAtoPictiteAll(BotList, rnnNNweights1, rnnMNweights1, 1.f, dendritecount, DepthBotPopulation, 0,
		constant.TopologyRNN.size(), constant.ArrayDim, constant.Topology, constant.TopologyRNN);
	graphGA.Display();


	for (int WGTick = 0; WGTick < constant.GlobalTicks; ++WGTick)
	{

		clock_t start2 = clock();
		
		InitBotList(BotList, Sector, constant);
		::memcpy(hBotList, BotList.data(), BotSize);
		hipMemcpy(dBotList, hBotList, BotSize, hipMemcpyHostToDevice);

		InitFoodList(FoodList, constant);
		::memcpy(hFoodList, FoodList.data(), FoodSize);
		hipMemcpy(dFoodList, hFoodList, FoodSize, hipMemcpyHostToDevice);

		InitAddFoodList(AddFoodList, constant);
		::memcpy(hAddFoodList, AddFoodList.data(), AddFoodSize);
		hipMemcpy(dAddFoodList, hAddFoodList, AddFoodSize, hipMemcpyHostToDevice);

		::memcpy(h_gbotSector, Sector.data(), sizeSensorElem);
		hipMemcpy(d_gbotSector, h_gbotSector, sizeSensorElem, hipMemcpyHostToDevice);
		::memcpy(h_tgbotSector, Sector.data(), sizeSensorElem);
		hipMemcpy(d_tgbotSector, h_tgbotSector, sizeSensorElem, hipMemcpyHostToDevice);
		float gameLevel = constant.gameLevel;

		
		clock_t finish2 = clock();

		for (int WTick = 0; WTick < constant.GAME_TICKS; ++WTick)
		{

			sf::Event event;
			if (graphModule.MainWindow.pollEvent(event))
			{
				if (event.type == sf::Event::Closed)
					graphModule.Close();

				if (event.type == sf::Event::KeyPressed)
					graphModule.KeyPressed(event);
			}


			//RNN Section

			clock_t ServerStart = clock();

		    cudaServer <<< int(1 + numElements / 32), 64 >> > (dBotList, dFoodList, d_gbotSector, d_cudaConst, numElements, WGTick);
			hipDeviceSynchronize();

			clock_t ServerFinish = clock();

            clock_t NNStart = clock();

			cudaBeforeNNRNN<<< int(1 + numElements / 32), 64 >>> (dBotList, d_gbotSector, d_tgbotSector, _argsRNN, d_cudaConst, numElements, WTick);
			hipDeviceSynchronize();

			cudaRNN <<< int(1 + numElements / 32), 64 >>> (dBotList, _argsRNN, d_cudaConst, d_Topology, d_TopologyRNN, numElements, WTick);
			hipDeviceSynchronize();

			cudaAfterNNRNN <<< int(1 + numElements / 32), 64 >>> (dBotList,d_gbotSector, _argsRNN, d_cudaConst, numElements, 
				WTick, d_fitnessIndex, d_fitnessValue);
			hipDeviceSynchronize();

            clock_t NNFinish = clock();
		
            clock_t start3 = clock();

			if (graphModule.DrawUnits)
			{
				

				// Copy the device result vector in device memory to the host result vector in host memory.
				//printf("Copy output data from the CUDA device to the host memory\n");

				hipMemcpy(hBotList, dBotList, BotSize, hipMemcpyDeviceToHost);
				hipMemcpy(hFoodList, dFoodList, FoodSize, hipMemcpyDeviceToHost);
				hipMemcpy(h_gbotSector,d_gbotSector, sizeSensorElem, hipMemcpyDeviceToHost);

				std::vector<Bot> VisualDepthBot(hBotList+ constant.BotPopulationSize*graphModule.DepthView, 
					 hBotList +constant.BotPopulationSize*graphModule.DepthView+ constant.BotPopulationSize);
				std::vector<Food> VisualDepthFood(hFoodList+ 4 * constant.FoodPopulationSize*graphModule.DepthView,
					hFoodList + 4 * constant.FoodPopulationSize*graphModule.DepthView+ 4 * constant.FoodPopulationSize);
				std::vector<float> VisualSector(h_gbotSector + 2*constant.numberofSensorshalf*constant.BotPopulationSize*graphModule.DepthView,
					h_gbotSector +2*constant.numberofSensorshalf*
					constant.BotPopulationSize*graphModule.DepthView + 2*constant.numberofSensorshalf* constant.BotPopulationSize);



			    std::memcpy(BotList.data(), hBotList, BotSize);
			
				
             


			   graphModule.Clear();
			   graphModule.DrawFrameInfo(WTick);
			   graphModule.DrawDepthAllBot(VisualDepthBot, VisualSector,constant.angleStep,
				   constant.angleStepInit, constant.numberofSensorshalf, constant.RadiusOfView);
			   graphModule.DrawDepthOther(VisualDepthFood, WTick);
			   graphModule.Display();

				constant.GAME_TICKS= graphModule.TickW;
				//constant.gamelevel = graphModule.gameLevel;

			}


			if (WTick % 50 == 0 && WTick > 15)
			{
				if (graphModule.DrawUnits == false)
					hipMemcpy(hFoodList, dFoodList, FoodSize, hipMemcpyDeviceToHost);

				for (int d = 0; d < constant.Depth; d++)
				{
					int FoodAddCount = 0;
					for (int i = 0; i < 4 * constant.FoodPopulationSize; i++)
					{
						int indexFoodDepth = i + d * 4 * constant.FoodPopulationSize;
						if (hFoodList[indexFoodDepth].Mass < 0)
						{
							int min = d * 4 * constant.FoodAddPopulationSize + 1;
							int max = d * 4 * constant.FoodAddPopulationSize + 4 * constant.FoodAddPopulationSize - 10;
							int output = min + (rand() % static_cast<int>(max - min));
							hFoodList[indexFoodDepth].Mass = AddFoodList[output].Mass;
							hFoodList[indexFoodDepth].X = AddFoodList[output].X;
							hFoodList[indexFoodDepth].Y = AddFoodList[output].Y;
							FoodAddCount++;
							if (FoodAddCount > 4 * AddFoodSize)
								break;
						}

					}
					
				}

				hipMemcpy(dFoodList, hFoodList, FoodSize, hipMemcpyHostToDevice);
			}
	
			clock_t finish3 = clock();

			infoGraph.Clear();
			infoGraph.TimetoPictite(constant.GAME_TICKS,
				(int)ceil(float(((ServerFinish - ServerStart) * 1000 / (CLOCKS_PER_SEC)))),
				(int)ceil(float(((NNFinish - NNStart) * 1000 / (CLOCKS_PER_SEC)))),
				(int)ceil(float(((finish2 - start2) * 1000 / (CLOCKS_PER_SEC)))),
				(int)ceil(float(((finish3 - start3) * 1000 / (CLOCKS_PER_SEC)))), WGTick, gameLevel);

			if (graphModule.DrawUnits)
			{
			//	infoGraph.BotStat(BotList, graphModule.DepthView, WGTick);
			}

			infoGraph.Display();


		}

			hipMemcpy(hBotList, dBotList, BotSize, hipMemcpyDeviceToHost);
			std::memcpy(BotList.data(), hBotList, BotSize);

			thrust::device_ptr<int> t_fitnessValue(d_fitnessValue);  // add this line before the sort line
			thrust::device_ptr<int> t_fitnessIndex(d_fitnessIndex);  // add this line before the sort line
																	 //thrust::sort(t_fitnessValue, t_fitnessValue + BotListSize);
																	 //srand(15);
			thrust::sort_by_key(t_fitnessValue, t_fitnessValue + DepthBotPopulation, t_fitnessIndex, thrust::greater<int>());

			hipMemcpy(h_fitnessValue, d_fitnessValue, sizeFitness, hipMemcpyDeviceToHost);
			hipMemcpy(h_fitnessIndex, d_fitnessIndex, sizeFitness, hipMemcpyDeviceToHost);
			std::vector<int> fitnessValue2(h_fitnessValue, h_fitnessValue + DepthBotPopulation);
			std::vector<int> fitnessIndex2(h_fitnessIndex, h_fitnessIndex + DepthBotPopulation);

			int sumFitness = thrust::reduce(t_fitnessValue, t_fitnessValue + DepthBotPopulation);


			int maxFitness = fitnessValue2[0];
			int minFitness = fitnessValue2[constant.DepthBotPopulation - 1];
			int bestBotindex = fitnessIndex2[0];
			int MiddleFitness = (int)((maxFitness + minFitness)/2);
			int MiddleFitness2 = (int)((fitnessValue2[25] + minFitness) / 2);
			int Fitness5 =(int)( maxFitness - (maxFitness - minFitness) /5);

			printf(" Sum fitness %u ", sumFitness);
			printf(" Best f0 %u ", fitnessValue2[0]);
			printf(" f1 %u ", fitnessValue2[1]);
			printf(" Mf %u ", MiddleFitness);
			printf(" Mf2 %u ", MiddleFitness2);
			printf(" F5 %u ", MiddleFitness2);
			printf(" lowest f %u ", fitnessValue2[constant.DepthBotPopulation - 1]);
			printf("\n");


		//	std::copy(BotListCopy.begin(), BotListCopy.end(), hBotListSort);
		//	std::memcpy(hBotListSort, BotListCopy.data(), BotSize);
		//	hipMemcpy(dBotListSort, hBotListSort, BotSize, hipMemcpyHostToDevice));

			//bestBotIndex = bestBotindex;

			//hiprandGenerator_t gen3r;
			//Set the generator options
			//hiprandCreateGenerator(&gen3r, HIPRAND_RNG_PSEUDO_DEFAULT);
			//Generate random numbers
			//hiprandSetPseudoRandomGeneratorSeed(gen3r, 1234ULL* WGTick);
			hiprandGenerateUniform(gen3r, drnnDNA, RnnDNAsize);
			//hiprandDestroyGenerator(gen3r);


			cudaRnd << < int(1 + RnnDNAsize / 32), 64 >> > (drnnDNA, RnnDNAsize);
			hipDeviceSynchronize();

			hipMemcpy(hrnnDNA, drnnDNA, rnnDNASize, hipMemcpyDeviceToHost);
			std::vector<float> rnnDNA2(hrnnDNA, hrnnDNA + RnnDNAsize);


			
/*
			cudaGARNN << < int(1 + numElements / 32), 64 >> > (dBotList, dBotListSort, drnnDNA, _argsRNN , d_Topology, d_TopologyRNN, preSizeMweight,
				d_cudaCardConst, numElements, WGTick, MiddleFitness, Fitness5, bestBotindex);
			hipDeviceSynchronize());
*/

			cudaGARNN2 << < int(1 + numElements / 32), 64 >> > (dBotList, d_fitnessIndex, drnnDNA, _argsRNN, d_Topology, d_TopologyRNN, preSizeMweight,
				d_cudaConst, numElements, WGTick, MiddleFitness2, Fitness5, sumFitness);
			hipDeviceSynchronize();

			initNNRNN << < int(1 + numElements / 32), 64 >> > (dBotList, _argsRNN, d_cudaConst, numElements, WGTick);
			hipDeviceSynchronize();

			//hipMemcpy(hBotList, dBotList, BotSize, hipMemcpyDeviceToHost));
			//std::memcpy(BotList.data(), hBotList, BotSize);



			if (graphModule.DrawUnits)
			{
				

				hipMemcpy(NNweights, _NNweights, sizeNweight, hipMemcpyDeviceToHost);
				std::vector<float> rnnNNweights(NNweights, NNweights + preSizeNweight);

				hipMemcpy(MNweights, _MNweights, sizeNweight, hipMemcpyDeviceToHost);
				std::vector<float> rnnMNweights(MNweights, MNweights + preSizeMweight);

				graphGA.Clear();
				graphGA.GAtoPictiteAll(BotList, rnnNNweights, rnnMNweights, 1.f, dendritecount, DepthBotPopulation, WGTick,
					constant.TopologyRNN.size(),constant.ArrayDim, constant.Topology, constant.TopologyRNN);
				graphGA.Display();
			}


		//RNN Block

		smallinitNNRNN << < int(1 + numElements / 32), 64 >> > (dBotList, _argsRNN, d_cudaConst, numElements, WGTick);
		hipDeviceSynchronize();

		hipMemcpy(hBotList, dBotList, BotSize, hipMemcpyDeviceToHost);
		std::memcpy(BotList.data(), hBotList, BotSize);

	}



	//resultOutput = MatCopy2d(ht2d, 3, 1);
	//resultCellState = MatCopy2d(ct2d, 3, 1);
	//return resultCudaOutput;



	hipFree(dBotList);
	hipFree(dFoodList);
	hipFree(dAddFoodList);

	hipFree(_outputs);
	hipFree(_sums);
	hipFree(_sumsContext);
	hipFree(_NNweights);
	hipFree(_MNweights);
	hipFree(_neuronContext);


	free(hBotList);
	free(hFoodList);
	free(hAddFoodList);

	free(outputs);
	free(sums);
	free(sumsContext);
	free(NNweights);
	free(MNweights);
	free(neuronContext);

}
